#include "hip/hip_runtime.h"

#include <iostream>
#include <stack>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "Complex.cu"
#include <GL/freeglut.h>

#define WINDOW_DIM 512
#define NUM_THREADS 32 

using namespace std;

Complex minC(-2.0, -1.2);
Complex maxC(1.0, 1.8);
Complex* dev_minC;
Complex* dev_maxC;
Complex* dev_c; 
int* dev_iteration; 

const int maxIt = 2000;

Complex* c = new Complex[WINDOW_DIM * WINDOW_DIM];
int iteration[WINDOW_DIM * WINDOW_DIM];  

struct Position 
{
  Position() : x(0), y(0) {}
  float x, y; 
};
Position start, end; 

struct Memory 
{
  Memory(float a, float b, float c, float d)
    : minC_r(a), minC_i(b), maxC_r(c), maxC_i(d) {}
  float minC_r, minC_i, maxC_r, maxC_i;
};
stack<Memory> memory_stack;

class RGB
{
public:
  RGB()
    : r(0), g(0), b(0) {}
  RGB(double r0, double g0, double b0)
    : r(r0), g(g0), b(b0) {}
public:
  double r;
  double g;
  double b;
};

RGB* colors = NULL;

void InitializeColors()
{
  colors = new RGB[maxIt + 1];
  for (int i = 0; i < maxIt; ++i)
    {
      if (i < 5)
        { 
          colors[i] = RGB(1, 1, 1);
        }
      else
        {
          colors[i] = RGB(drand48(), drand48(), drand48());
        }
    }
  colors[maxIt] = RGB(); 
}

__global__ void ComputeIteration(Complex* dev_minC, Complex* dev_maxC, int* dev_iteration, Complex* dev_c) 
{
  int id = threadIdx.x + blockIdx.x * blockDim.x; 
  int i = id / WINDOW_DIM;   
  int j = id % WINDOW_DIM;
  
  double dX = dev_maxC->r - dev_minC->r;
  double dY = dev_maxC->i - dev_minC->i;
  double nX = (double) i / WINDOW_DIM;
  double nY = (double) j / WINDOW_DIM;
  
  dev_c[id].r = dev_minC->r + nX * dX;
  dev_c[id].i = dev_minC->i + nY * dY;

  Complex Z;
  Z.r = dev_c[id].r;
  Z.i = dev_c[id].i;
  dev_iteration[id] = 0;
      
  while(dev_iteration[id] < 2000 && Z.magnitude2() < 4.0)
  {
    dev_iteration[id]++;
    Z = (Z*Z) + dev_c[id];
  }
}

void ComputeMBSet() 
{
  hipMalloc((void**)&dev_iteration, WINDOW_DIM * WINDOW_DIM * sizeof(int));
  hipMalloc((void**)&dev_minC, sizeof(Complex));
  hipMalloc((void**)&dev_maxC, sizeof(Complex));
  hipMalloc((void**)&dev_c, WINDOW_DIM * WINDOW_DIM * sizeof(Complex));
  hipMemcpy(dev_minC, &minC, sizeof(Complex), hipMemcpyHostToDevice);
  hipMemcpy(dev_maxC, &maxC, sizeof(Complex), hipMemcpyHostToDevice);
  hipMemcpy(dev_iteration, iteration, WINDOW_DIM * WINDOW_DIM * sizeof(int), hipMemcpyHostToDevice);  
  hipMemcpy(dev_c, c, WINDOW_DIM * WINDOW_DIM * sizeof(Complex), hipMemcpyHostToDevice);  
  
  ComputeIteration<<< WINDOW_DIM * WINDOW_DIM / NUM_THREADS, NUM_THREADS >>>(dev_minC, dev_maxC, dev_iteration, dev_c);

  hipMemcpy(iteration, dev_iteration, WINDOW_DIM * WINDOW_DIM * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(c, dev_c, WINDOW_DIM * WINDOW_DIM * sizeof(Complex), hipMemcpyDeviceToHost);  
}

void display(void)  
{
  glClearColor(0.0, 0.0, 0.0, 0.0); 
  glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
  glLoadIdentity();
  
  glBegin(GL_POINTS);
  for(int i = 0; i < WINDOW_DIM; i++)
  {
    for(int j = 0; j < WINDOW_DIM; j++)
    {
      glColor3f(colors[iteration[i*WINDOW_DIM + j]].r, colors[iteration[i*WINDOW_DIM + j]].g, colors[iteration[i*WINDOW_DIM + j]].b);
      glVertex2d(i, j);
    }
  }
  glEnd();

  glutSwapBuffers();
}

void keyboard (unsigned char key, int x, int y) 
{
  if(key == 'q') 
  { 
    exit(0);  
  }

  if(key == 'b')
  {
    if(memory_stack.size() > 0)
    {
      Memory temp = memory_stack.top();  
      memory_stack.pop();          
      minC.r = temp.minC_r;
      minC.i = temp.minC_i;
      maxC.r = temp.maxC_r;
      maxC.i = temp.maxC_i;
      ComputeMBSet();   
      glutPostRedisplay();
    }
    else
      cout<<"You cannot go back any more!"<<endl;
  }
}

void mouse(int button, int state, int x, int y) 
{
  if(button == GLUT_LEFT_BUTTON && state == GLUT_DOWN) 
  {
    start.x = x;
    start.y = y;
  }

  if(button == GLUT_LEFT_BUTTON && state == GLUT_UP)                      
  {
    memory_stack.push(Memory(minC.r, minC.i, maxC.r, maxC.i));   
    
    double dx = abs(x - start.x);
    double dy = abs(y - start.y);
    double ds = dx > dy ? dy : dx;

    end.x = x > start.x ? start.x + ds : start.x - ds;
    end.y = y > start.y ? start.y + ds : start.y - ds;

    int min_i = min(start.x, end.x), min_j = min(start.y, end.y);
    minC.r = c[min_i*WINDOW_DIM + min_j].r;
    minC.i = c[min_i*WINDOW_DIM + min_j].i;
    
    int max_i = max(start.x, end.x), max_j = max(start.y, end.y);
    maxC.r = c[max_i*WINDOW_DIM + max_j].r;
    maxC.i = c[max_i*WINDOW_DIM + max_j].i;

    ComputeMBSet();  
    glutPostRedisplay(); 
  }
}

int main(int argc, char** argv)
{
  glutInit(&argc, argv);
  glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA | GLUT_DEPTH);
  glutInitWindowSize(WINDOW_DIM, WINDOW_DIM);
  glutCreateWindow("MBSet");

  glViewport(0, 0, WINDOW_DIM, WINDOW_DIM);                                            
  glMatrixMode(GL_PROJECTION); 
  glLoadIdentity();

  gluOrtho2D(0, WINDOW_DIM, WINDOW_DIM, 0);
  glMatrixMode(GL_MODELVIEW);
  glLoadIdentity();  
  
  InitializeColors();
  ComputeMBSet();
  
  glutDisplayFunc(display);
  glutIdleFunc(display);
  glutKeyboardFunc (keyboard);
  glutMouseFunc(mouse);

  glutMainLoop();
  return 0;
}
